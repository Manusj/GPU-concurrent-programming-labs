#include "hip/hip_runtime.h"
// Lab 5, image filters with CUDA.

// Compile with a command-line similar to Lab 4:
// nvcc filter.cu -c -arch=sm_30 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -lcudart -L/usr/local/cuda/lib -lglut -o filter
// or (multicore lab)
// nvcc filter.cu -c -arch=sm_20 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -L/usr/local/cuda/lib64 -lcudart -lglut -o filter

// 2017-11-27: Early pre-release, dubbed "beta".
// 2017-12-03: First official version! Brand new lab 5 based on the old lab 6.
// Better variable names, better prepared for some lab tasks. More changes may come
// but I call this version 1.0b2.
// 2017-12-04: Two fixes: Added command-lines (above), fixed a bug in computeImages
// that allocated too much memory. b3
// 2017-12-04: More fixes: Tightened up the kernel with edge clamping.
// Less code, nicer result (no borders). Cleaned up some messed up X and Y. b4
// 2022-12-07: A correction for a deprecated function.

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#ifdef __APPLE__
  #include <GLUT/glut.h>
  #include <OpenGL/gl.h>
#else
  #include <GL/glut.h>
#endif
#include "readppm.h"
#include "milli.h"

// Use these for setting shared memory size.
#define maxKernelSizeX 10
#define maxKernelSizeY 10
#define blocksize 16
#define image_size 512

hipEvent_t startEvent;
hipEvent_t endEvent;

__global__ void filter(unsigned char *image, unsigned char *out, const unsigned int imagesizex, const unsigned int imagesizey, const int kernelsizex, const int kernelsizey)
{ 
  // map from blockIdx to pixel position
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

  int dy, dx;
  unsigned int sumx, sumy, sumz;

//initilize shared memory	TODO
	__shared__ unsigned char shared_mem[(blocksize+2*maxKernelSizeX+1)*(blocksize+2*maxKernelSizeY+1)*3];//but dimension is wrong maybe go for imagex/gridx * imagey/gridy *3 but we need those from input
	//for now we keep it as max size anyway
	//shared mem is same as block + kernel
	//start of our filling set blockIdx.x * blockDim.x  this is output 

	//start of our data read set would be blockIdx.x * blockDim.x  - kernelx this is input
	//write same for y		watch out for x3, for now we say it doesnt matter
	unsigned int my_shared_mem_index_x = threadIdx.x;
	unsigned int my_shared_mem_index_y = threadIdx.y;
	//for(dy=-kernelsizey;dy<=kernelsizey;dy++)
		//for(dx=-kernelsizex;dx<=kernelsizex;dx++)
	if((my_shared_mem_index_x) % ((2*kernelsizex)+1)==0)
		if((my_shared_mem_index_y) % ((2*kernelsizey)+1)==0)
	for(dy=-kernelsizey;dy<=kernelsizey;dy++)
		for(dx=-kernelsizex;dx<=kernelsizex;dx++)
		{
			int yy = min(max(y+dy, 0), imagesizey-1);//this is border
			int xx = min(max(x+dx, 0), imagesizex-1);//this is border

			int shared_index_x= min(max(my_shared_mem_index_x+dx, 0), blocksize+2*kernelsizex);
			int shared_index_y= min(max(my_shared_mem_index_y+dy, 0), blocksize+2*kernelsizey);
			shared_mem[(shared_index_x+((shared_index_y)*(blocksize+2*kernelsizex)))*3+0] = image[((yy)*imagesizex+(xx))*3+0];		//now these change
			shared_mem[(shared_index_x+((shared_index_y)*(blocksize+2*kernelsizex)))*3+1] = image[((yy)*imagesizex+(xx))*3+1];		//TODO
			shared_mem[(shared_index_x+((shared_index_y)*(blocksize+2*kernelsizex)))*3+2] = image[((yy)*imagesizex+(xx))*3+2];
		}

	//synchronize
	__syncthreads();
	//done
	//if(threadIdx.x==0&&blockIdx.x==0&&threadIdx.y==0&&blockIdx.y==0)
	//	printf("boop kernelsizex = %d \n",kernelsizex);

  int divby = (2*kernelsizex+1)*(2*kernelsizey+1); // Works for box filters only!
	
	if (x < imagesizex && y < imagesizey) // If inside image
	{
// Filter kernel (simple box filter)
	sumx=0;sumy=0;sumz=0;
	for(dy=-kernelsizey;dy<=kernelsizey;dy++)
	{
		for(dx=-kernelsizex;dx<=kernelsizex;dx++)	
		{
			// Use max and min to avoid branching!
			int yy = min(max(y+dy, 0), imagesizey-1);//this is border
			int xx = min(max(x+dx, 0), imagesizex-1);//this is border
			
			//int shared_index_x= min(max(x+dx, 0), blocksize+2*kernelsizex-1);
			//int shared_index_y= min(max(y+dy, 0), blocksize+2*kernelsizey-1);
			int shared_index_x= min(max(my_shared_mem_index_x+dx, 0), blocksize+2*kernelsizex);
			int shared_index_y= min(max(my_shared_mem_index_y+dy, 0), blocksize+2*kernelsizey);
			sumx += shared_mem[(shared_index_x+((shared_index_y)*(blocksize+2*kernelsizex)))*3+0];		//now these change
			sumy += shared_mem[(shared_index_x+((shared_index_y)*(blocksize+2*kernelsizex)))*3+1];		//TODO
			sumz += shared_mem[(shared_index_x+((shared_index_y)*(blocksize+2*kernelsizex)))*3+2];
			
		
			//sumx += image[((yy)*imagesizex+(xx))*3+0];
			//sumy += image[((yy)*imagesizex+(xx))*3+1];
			//sumz += image[((yy)*imagesizex+(xx))*3+2];
		
		/*
			//if(my_shared_mem_index_x>20&&my_shared_mem_index_y>20)
			{
			out[(y*imagesizex+x)*3+0] = shared_mem[(shared_index_x+((shared_index_y)*(blocksize+2*kernelsizex)))*3+0];
			out[(y*imagesizex+x)*3+1] = shared_mem[(shared_index_x+((shared_index_y)*(blocksize+2*kernelsizex)))*3+1];
			out[(y*imagesizex+x)*3+2] = shared_mem[(shared_index_x+((shared_index_y)*(blocksize+2*kernelsizex)))*3+2];
			}
		*/
		}
	}
	//if(threadIdx.x==0&&blockIdx.x==0&&threadIdx.y==0&&blockIdx.y==0)
	//	printf("sumy= %u \n",sumy);
	//if(threadIdx.x==0&&blockIdx.x==0&&threadIdx.y==0&&blockIdx.y==0)
	//	printf("divby= %d \n",divby);
	out[(y*imagesizex+x)*3+0] = sumx/divby;
	out[(y*imagesizex+x)*3+1] = sumy/divby;
	out[(y*imagesizex+x)*3+2] = sumz/divby;
	//if(threadIdx.x==0&&blockIdx.x==0&&threadIdx.y==0&&blockIdx.y==0)
	//	printf("out= %uc \n",out[(y*imagesizex+x)*3+1]);

	}
}

// Global variables for image data

unsigned char *image, *pixels, *dev_bitmap, *dev_input;
unsigned int imagesizey, imagesizex; // Image size

////////////////////////////////////////////////////////////////////////////////
// main computation function
////////////////////////////////////////////////////////////////////////////////
void computeImages(int kernelsizex, int kernelsizey)
{
	if (kernelsizex > maxKernelSizeX || kernelsizey > maxKernelSizeY)
	{
		printf("Kernel size out of bounds!\n");
		return;
	}
	float timediff;
	hipEventCreate(&startEvent);
	hipEventCreate(&endEvent);

	pixels = (unsigned char *) malloc(imagesizex*imagesizey*3);
	hipMalloc( (void**)&dev_input, imagesizex*imagesizey*3);
	hipMemcpy( dev_input, image, imagesizey*imagesizex*3, hipMemcpyHostToDevice );
	hipMalloc( (void**)&dev_bitmap, imagesizex*imagesizey*3);
	dim3 block(blocksize,blocksize);		//preferably 32
	dim3 grid(imagesizex/blocksize,imagesizey/blocksize);		//should change to imagesize/blocksize
	hipEventSynchronize(startEvent);
	hipEventRecord(startEvent, 0);
	filter<<<grid,block>>>(dev_input, dev_bitmap, imagesizex, imagesizey, kernelsizex, kernelsizey); // Awful load balance
	hipDeviceSynchronize();
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timediff, startEvent, endEvent);
	printf("timediff - %lf\n",timediff);
//	Check for errors!
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
	hipMemcpy( pixels, dev_bitmap, imagesizey*imagesizex*3, hipMemcpyDeviceToHost );
	hipFree( dev_bitmap );
	hipFree( dev_input );
}

// Display images
void Draw()
{
// Dump the whole picture onto the screen.	
	glClearColor( 0.0, 0.0, 0.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT );

	if (imagesizey >= imagesizex)
	{ // Not wide - probably square. Original left, result right.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
		glRasterPos2i(0, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE,  pixels);
	}
	else
	{ // Wide image! Original on top, result below.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, pixels );
		glRasterPos2i(-1, 0);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
	}
	glFlush();
}

// Main program, inits
int main( int argc, char** argv) 
{
	glutInit(&argc, argv);
	glutInitDisplayMode( GLUT_SINGLE | GLUT_RGBA );

	if (argc > 1)
		image = readppm(argv[1], (int *)&imagesizex, (int *)&imagesizey);
	else
		image = readppm((char *)"maskros512.ppm", (int *)&imagesizex, (int *)&imagesizey);

	if (imagesizey >= imagesizex)
		glutInitWindowSize( imagesizex*2, imagesizey );
	else
		glutInitWindowSize( imagesizex, imagesizey*2 );
	glutCreateWindow("Lab 5");
	glutDisplayFunc(Draw);

	ResetMilli();

	computeImages(2, 2);

// You can save the result to a file like this:
//	writeppm("out.ppm", imagesizey, imagesizex, pixels);

	glutMainLoop();
	return 0;
}
