// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.
// Update 2022: Changed to cudaDeviceSynchronize.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
const int N = 64; 
const int blocksize = 32; 
const int gridsize = 2;
hipEvent_t startEvent;
hipEvent_t endEvent;

__global__ 
void simple(float *c) 
{
	float x = c[threadIdx.x];
	float qrt = sqrt(x);
	c[threadIdx.x] = qrt;
}

__global__ 
void add_matrix(float *a, float *b, float *c, int N)
{
	int index;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

	index = i + j*N;
	c[index] = a[index] + b[index];
}

int main()
{
	float *a = new float[N*N];	
    float *b = new float[N*N];
    float *c = new float[N*N];		
	float *dest;
    float *a_gpu;
    float *b_gpu;
	float timediff;
	const int size = N*N*sizeof(float);
	hipEventCreate(&startEvent);
	hipEventCreate(&endEvent);
	
	

	hipMalloc( (void**)&dest, size);
    hipMalloc( (void**)&a_gpu, size);
    hipMalloc( (void**)&b_gpu, size);

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}
	

	dim3 dimBlock( blocksize, blocksize );
	dim3 dimGrid( gridsize, gridsize );
	hipMemcpy( a_gpu, a, size, hipMemcpyHostToDevice ); 
    hipMemcpy( b_gpu, b, size, hipMemcpyHostToDevice ); 
	hipEventSynchronize(startEvent);
	hipEventRecord(startEvent, 0);
	add_matrix<<<dimGrid, dimBlock>>>(a_gpu, b_gpu, dest, N);
	hipDeviceSynchronize();
	hipEventSynchronize(endEvent);
	hipEventRecord(endEvent, 0);
	hipEventElapsedTime(&timediff, startEvent, endEvent);
	
	hipMemcpy( c, dest, size, hipMemcpyDeviceToHost ); 
	hipFree( dest );
    hipFree( a_gpu );
    hipFree( b_gpu );
	
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%0.2f ", c[i+j*N]);
		}
		printf("\n");
	}
	printf("\n");
	delete[] a;
    delete[] b;
    delete[] c;
	printf("done\n");
	printf("timediff - %lf\n");
	return EXIT_SUCCESS;
}

/* Best Gpu performance
const int N = 32; 
const int blocksize = 32; 
const int gridsize = 1;
*/
