// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.
// Update 2022: Changed to cudaDeviceSynchronize.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
const int N = 10; 
const int blocksize = 10; 

__global__ 
void simple(float *c) 
{
	float x = c[threadIdx.x];
	float qrt = sqrt(x);
	c[threadIdx.x] = qrt;
}

int main()
{
	float *c = new float[N];	
	float *cd;
	const int size = N*sizeof(float);
	
	

	hipMalloc( (void**)&cd, size );

	for(int i =1; i<N;i++)
	{
		c[i] = i;
		printf("-%f-",sqrt(c[i]));
	}
	printf("\n");
	

	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hipMemcpy( cd, c, size, hipMemcpyHostToDevice ); 
	simple<<<dimGrid, dimBlock>>>(cd);
	hipDeviceSynchronize();
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 
	hipFree( cd );
	
	for (int i = 0; i < N; i++)
		printf("%f ", c[i]);
	printf("\n");
	delete[] c;
	printf("done\n");
	return EXIT_SUCCESS;
}
